#include "hip/hip_runtime.h"
#include "src/gpu_utils/cuda_lasso.cuh"
#include <iomanip>
#include <signal.h>
#include <queue>
#include "src/fftw.h"
#include <math.h>
#include "src/gpu_utils/cuda_utils_cub.cuh"
#include "src/gpu_utils/cuda_fft.h"
#include "src/gpu_utils/cuda_kernels/helper.cuh"

inline int mapToCompact(int k, int i, int j, int Z, int Y, int X, int ZZ, int YY, int XX){
    if(k >= ZZ/2) k += ZZ;
    if(i >= YY/2) i += YY;
    if(j >= XX/2) j += XX;
    return k*Y*X + i*X + j;
}

void cuda_lasso(int tv_iters, RFLOAT l_r, RFLOAT mu, RFLOAT tv_alpha, RFLOAT tv_beta, RFLOAT eps, MultidimArray<RFLOAT> &Fconv,
        MultidimArray<RFLOAT> &Fweight, MultidimArray<RFLOAT> &vol_out, MlDeviceBundle *devBundle, int data_dim, RFLOAT normalise, RFLOAT nrparts, bool do_nag, RFLOAT implicit_weight){
    //normfft = max(normfft, 1.);
    hipSetDevice(devBundle->device_id);
    devBundle->setStream();
    std::cout <<" Device: " << devBundle->device_id <<", " << data_dim;
    std::cout << " " << vol_out.xdim << ", " << vol_out.ydim << ", " << vol_out.zdim << std::endl;
    //std::cout << "tv_iters: " << tv_iters << " ";
    int ZZ = vol_out.zdim >> 1;
    int YY = vol_out.ydim >> 1;
    int XX = vol_out.xdim >> 1;
    int Z = vol_out.zdim;
    int Y = vol_out.ydim;
    int X = vol_out.xdim;

    int img_size = vol_out.nzyxdim;
    int img_size_h = 0;

    if(data_dim == 3) img_size_h = img_size/8;
    else img_size_h = img_size/4;

    std::priority_queue<XFLOAT, std::vector<XFLOAT>, std::greater<XFLOAT>> pq;
    CudaUnifedPtr<XFLOAT> img(devBundle->stream, devBundle->device_id, img_size);
    //momentum term
    //CudaUnifedPtr<XFLOAT> momentum(devBundle->stream, devBundle->device_id, img_size);
    CudaUnifedPtr<XFLOAT> yob(devBundle->stream, devBundle->device_id, img_size);
    CudaUnifedPtr<XFLOAT> grads(devBundle->stream, devBundle->device_id, img_size);
    //CudaUnifedPtr<XFLOAT> vol(devBundle->stream, devBundle->device_id, img_size);
    CudaUnifedPtr<XFLOAT> sigma_norm(devBundle->stream, devBundle->device_id, 1);
    XFLOAT mem_size = vol_out.getSize()*sizeof(XFLOAT)/1024./1024.;
    //std::cout << "Mem size " << 4*mem_size << " " << 3*img_size_h*sizeof(XFLOAT)/1024./1024. << "MB "<< std::endl;
    //CudaFFTT<false> transformer(devBundle->stream, NULL, data_dim);
    CudaFFTU transformer(devBundle->stream, devBundle->device_id, data_dim);
    transformer.setSize(vol_out.xdim, vol_out.ydim, vol_out.zdim);
    CudaUnifedPtr<XFLOAT> weight(devBundle->stream, devBundle->device_id, Fweight.getSize());
    //img.device_alloc();
    img.setPtr(transformer.reals.ptr);
    //img.alloc();
    //momentum is used to calculate grads
    //momentum.setPtr(transformer.reals.ptr);
    yob.alloc();
    //vol.alloc();
    sigma_norm.alloc();
    XFLOAT lambda = implicit_weight*normalise;
    std::cout << "weight " << weight.getSize() << ", " << Fconv.getSize()<< std::endl;
    RFLOAT fconv_norm = 0.;
    XFLOAT yob_norm = 0.;
    int median_size = img_size_h/128;
    int sparse_count = 0;

    if(do_nag) {
        for(int i = 0; i < Fconv.nzyxdim; i++){
            //img[i] = 0.f;
            //transformer.fouriers[i].x = Fconv.data[i].real;
            //transformer.fouriers[i].y = Fconv.data[i].imag;
            yob[i] = Fconv.data[i];
            fconv_norm += Fconv.data[i]*Fconv.data[i];
            yob[i] += lambda*((XFLOAT)vol_out.data[i]);
            //vol[i] = lambda*((XFLOAT)vol_out.data[i]);
            //std::cout << yob[i] << std::endl;
        }
        fconv_norm = sqrt(fconv_norm/Fconv.nzyxdim);
        //for(int k = 0; k < ZZ; k++)
        //    for(int i = 0; i < YY; i++)
        //        for(int j = 0; j < XX; j++){
        //            int index = mapToCompact(k, i, j, Z, Y, X, ZZ, YY, XX);
        //            int my_index = k*YY*XX + i*XX + j;
        //            yob[my_index] = Fconv.data[index];
        //            yob[my_index] += lambda*((XFLOAT)vol_out.data[index]);
        //        }
        //for(int i = 0; i < img_size; i++){
        //    img[i] = vol_out.data[i];
        //    //momentum[i] = momentum_out.data[i];
        //}
        //momentum.device_init(0.f);
        //vol_out is of even size
        FOR_ALL_DIRECT_ELEMENTS_IN_ARRAY3D(vol_out){
            if((k < ZZ || k >= Z - ZZ) &&
               (i < YY || i >= Y - YY) &&
               (j < XX || j >= X - XX)){
                img[k*Y*X + i*X + j] = DIRECT_A3D_ELEM(vol_out, k, i, j);
                if(img[k*Y*X + i*X + j] > 1.e-3) {
                    sparse_count += 1;
                    yob_norm += abs(img[k*Y*X + i*X + j]);
                } else {
                    img[k*Y*X + i*X + j] = 0.;
                }
                if(pq.size() < median_size) pq.push(img[k*Y*X + i*X + j]);
                else if(pq.top() < img[k*Y*X + i*X + j]) {
                    pq.pop();
                    pq.push(img[k*Y*X + i*X + j]);
                }
            } else {
                img[k*Y*X + i*X + j] = 0.;
            }
            //momentum[k*Y*X + i*X + j] = img[k*Y*X + i*X + j];
        }
        //img.init();
    } else {
        for(int i = 0; i < Fconv.nzyxdim; i++){
            //img[i] = 0.f;
            //transformer.fouriers[i].x = Fconv.data[i].real;
            //transformer.fouriers[i].y = Fconv.data[i].imag;
            yob[i] = Fconv.data[i];
            //std::cout << yob[i] << std::endl;
        }
        img.init();
        //momentum.device_init(0.f);
    }

    //adjust median according to sparseness count
    if(sparse_count*0.5 < median_size){
        int new_median_size = sparse_count*0.5;
        for(int i = 0; i < median_size - new_median_size; i++){
            pq.pop();
        }
    }
    std::cout << "median: " << pq.top() << " pq_fraction: " << pq.size()/XFLOAT(img_size_h) << " sparseness: " << sparse_count/XFLOAT(img_size_h) << std::endl;
    //yob.cp_to_device();
    yob.set_read_only();
    yob.attach_to_stream();
    //vol.set_read_only();
    //vol.attach_to_stream();
    img.attach_to_stream();
    transformer.fouriers.attach_to_device();
    transformer.fouriers.attach_to_stream();
    //yob.attach_to_device();
    //transformer.fouriers.cp_to_device();
    //transformer.backward(yob);
    //img.cp_to_device();
    //yob.dump_device_to_file("yob"+devBundle->device_id);
    
    grads.alloc();
    grads.attach_to_stream();
    //move Fweight from host to device
    size_t free_byte ;
    size_t total_byte ;
    //hipMemGetInfo( &free_byte, &total_byte ) ;
    //std::cout << "Free Mem " << free_byte/1024./1024. << " Total Mem " << total_byte/1024./1024. << std::endl;

    weight.alloc();
    //int xdim = Fconv.xdim/2 + 1;
    XFLOAT max_weight = 0;
    XFLOAT min_weight = normalise;
    FOR_ALL_ELEMENTS_IN_FFTW_TRANSFORM(Fweight){
        int kw = kp;
        int iw = ip;
        int jw = jp;
        if(kp < 0) kw += Fweight.zdim;
        if(ip < 0) iw += Fweight.ydim;
        int index = kw*Fweight.ydim*Fweight.xdim + iw*Fweight.xdim + jw;
        weight[index] = FFTW_ELEM(Fweight, kp, ip, jp);
        if(max_weight < weight[index]) max_weight = weight[index];
        if(min_weight > weight[index] && weight[index] > 1e-5) min_weight = weight[index];        
        //std::cout << weight[index] << " " << index << std::endl;
    }
    //for(int i = 0; i < Fweight.nzyxdim; i++){
    //    weight[i] = Fweight.data[i];
    //    tot_weight += std::abs(weight[i]);
    //}
    weight.set_read_only();
    weight.attach_to_stream();
    //synchronize before launch kernel
    img.streamSync();
    weight.cp_to_device();
    //std::cout << ~weight << " " << ~grads << " " << ~img << " " << ~yob << std::endl;
    yob_norm /= sparse_count;
    //eps = yob_norm*1.;
    eps = max(pq.top(), 0.1);
    XFLOAT tv_eps = 0.00005;
    XFLOAT tv_log_eps = eps/3;
    int FBsize = (int) ceilf((float)transformer.fouriers.getSize()/(float)BLOCK_SIZE);
    int imgBsize = (int) ceilf((float)img_size_h/(float)BLOCK_SIZE);
    int imgBFsize = (int) ceilf((float)img.getSize()/(float)BLOCK_SIZE);
    //cuda_kernel_multi<<<imgBsize, BLOCK_SIZE, 0, yob.getStream()>>>(
    //            ~yob,
    //            (XFLOAT)1/((XFLOAT)normfft),
    //            yob.getSize());
    XFLOAT weight_norm = getSquareSumOnBlock(weight);
    weight_norm = sqrt(weight_norm/weight.getSize());
    
    //RFLOAT yob_norm = getMaxOnDevice(yob);
    //yob_norm = sqrt(yob_norm/img_size);
    std::cout << "device: " << devBundle->device_id << " weight: " << weight_norm << std::endl;
    Fweight.printShape();
    //l_r *= 1./(weight_norm);
    //std::cout << "start optimizing " << "yob: " << yob_norm << ", " << "weight: " << weight_norm << ", l_r: " << l_r << ", " << std::endl;
    l_r = l_r/(max_weight + normalise);
    std::cout << "start optimizing " << "lr: " << l_r << " lambda: " << lambda << " avg weight : " << normalise << " max weight: " << max_weight << " min weight: " << min_weight << " condition number: " << max_weight/min_weight << std::endl;
    tv_alpha *= std::sqrt(normalise);
    tv_beta *= std::sqrt(normalise);
    //XFLOAT yob_norm = getSquareSumOnBlock(img);
    //XFLOAT yob_norm = 0;
    //for(int i = 0; i < img_size; i++){
    //    yob_norm += img[i]*img[i];
    //}
    std::cout << "tv_alpha: " << tv_alpha << ", tv_beta: " << tv_beta << " fconv_norm: " << fconv_norm << " img_norm: " << yob_norm <<" img_size " << img_size << " FBsize " << FBsize << std::endl;
    XFLOAT scale = (XFLOAT)1/((XFLOAT)transformer.reals.getSize());
    XFLOAT w = tv_alpha/eps*0.5;
    XFLOAT tk = 1.;
    for(int m_c = 0; m_c <= tv_iters; m_c++){
        //forward transform img/momentum
        transformer.forward();
        //multiply with weight and normalization factor
        //cuda_kernel_complex_multi<<<FBsize, BLOCK_SIZE, 0, transformer.fouriers.getStream()>>>(
        //        (XFLOAT*)~transformer.fouriers,
        //        ~weight,
        //        (XFLOAT)1/((XFLOAT)transformer.reals.getSize()),
        //        transformer.fouriers.getSize());
        int xdim = X/2 + 1;
        cuda_kernel_complex_multi<<<FBsize, BLOCK_SIZE, 0, transformer.fouriers.getStream()>>>(
                (XFLOAT*)~transformer.fouriers,
                ~weight,
                scale,
                w*scale,
                Z,
                Y,
                xdim,
                Fweight.zdim,
                Fweight.ydim,
                Fweight.xdim,
                transformer.fouriers.getSize());

        //inverse transform and put the transformation on grads
        transformer.backward(grads);
        //transformer.reals.cp_on_device(grads.d_ptr);
        //substract Mout
        //cuda_kernel_substract<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
        //        ~grads,
        //        ~yob,
        //        grads.getSize());
        //work on windowed grads
        //sigma_norm[0] = 0.;
        cuda_kernel_substract<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                ~grads,
                ~yob,
                ~img,
                //~momentum,
                //~vol,
                (XFLOAT)lambda,
                //~sigma_norm,
                ZZ,
                YY,
                XX,
                img_size_h);
                //grads.getSize());
        //synchronize before returning sum
        //sigma_norm.streamSync();
        //sigma_norm[0] = sqrt(sigma_norm[0]/img_size_h);
        //get graph gradient
        if(data_dim == 3){
            //work on windowed grads
            cuda_kernel_graph_grad<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                    ~img,
                    //~momentum,
                    ~grads,
                    Z,
                    Y,
                    X,
                    ZZ,
                    YY,
                    XX,
                    tv_beta,
                    tv_log_eps,//eps of log approximation
                    tv_eps,//eps of l1 norm approximation
                    //grads.getSize());
                    img_size_h);
        } else {
            cuda_kernel_graph_grad<<<imgBFsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                    ~img,
                    ~grads,
                    Y,
                    X,
                    tv_beta,
                    tv_eps,
                    grads.getSize());

        }
        //update momentum
        //cuda_kernel_update_momentum<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
        //        ~grads,
        //        ~momentum,
        //        mu,
        //        l_r,
        //        grads.getSize());
        //update momentum
        XFLOAT tk1 = (1. + sqrt(1. + 4.*tk*tk))/2.;
        mu = (tk - 1)/tk1;
        tk = tk1;
        cuda_kernel_soft_threshold<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                ~img,
                //~momentum,
                ~grads,
                //mu,
                l_r,
                tv_alpha,
                eps,
                X,
                Y,
                Z,
                XX,
                YY,
                ZZ,
                //grads.getSize());
                img_size_h);
        //update momentum and soft thresholding
        //cuda_kernel_soft_threshold<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
        //        ~img,
        //        ~momentum,
        //        ~grads,
        //        ZZ,
        //        YY,
        //        XX,
        //        mu,
        //        l_r,
        //        tv_alpha,
        //        eps,
        //        momentum.getSize());
        if(m_c % 100 == 0){
            std::cout << m_c << " ";
            //std::cout << sigma_norm[0] << " ";
            RFLOAT img_norm = getSquareSumOnBlock(img);
            RFLOAT resi_norm = getSquareSumOnBlock(grads);
            img_norm = sqrt(img_norm/img_size_h);
            resi_norm = sqrt(resi_norm/img_size_h);
            std::cout <<  resi_norm << " " << img_norm << " " << resi_norm/img_norm << std::endl;
        }
        //l_r *= exp(-0.0025);
    }
    //now copy image to host
    //now set vol_out and wait for stream to complete
    img.streamSync();
    bool hasnan = false;
    for(int i = 0; i < img_size; i++){
        if(isnan(img[i])){
            vol_out.data[i] = 0.;
            hasnan = true;
        }
        else
        {
            //consider upscale image
            vol_out.data[i] = (1. + w/img_size_h)*img[i];
        }
    }
    if(hasnan) std::cout << "WARNING: find nan in reconstruction." << std::endl;
    transformer.clear();
    devBundle->destroyStream();
}

//void cuda_lasso(int tv_iters, RFLOAT l_r, RFLOAT mu, RFLOAT tv_alpha, RFLOAT tv_beta, RFLOAT eps, MultidimArray<Complex> &Fconv,
//        MultidimArray<RFLOAT> &Fweight, MultidimArray<RFLOAT> &vol_out, MlDeviceBundle *devBundle, int data_dim, RFLOAT normfft, bool do_nag){
//    //normfft = max(normfft, 1.);
//    hipSetDevice(devBundle->device_id);
//    devBundle->setStream();
//    std::cout <<" Device: " << devBundle->device_id <<", " << data_dim;
//    std::cout << " " << vol_out.xdim << ", " << vol_out.ydim << ", " << vol_out.zdim << std::endl;
//    int img_size = vol_out.nzyxdim;
//    CudaGlobalPtr<XFLOAT, false> img(img_size, devBundle->stream);
//    CudaGlobalPtr<XFLOAT, false> yob(img_size, devBundle->stream);
//    CudaGlobalPtr<XFLOAT, false> weight(Fweight.nzyxdim, devBundle->stream);
//    CudaGlobalPtr<XFLOAT, false> grads(img_size, devBundle->stream);
//    CudaGlobalPtr<XFLOAT, false> momentum(img_size, devBundle->stream);
//    CudaFFTT<false> transformer(devBundle->stream, NULL, data_dim);
//    transformer.setSize(vol_out.xdim, vol_out.ydim, vol_out.zdim);
//    img.device_alloc();
//    yob.device_alloc();
//    std::cout << ", " << img_size << ", " << Fconv.getSize()<< std::endl;
//    for(int i = 0; i < Fconv.nzyxdim; i++){
//        //img[i] = 0.f;
//        transformer.fouriers[i].x = Fconv.data[i].real;
//        transformer.fouriers[i].y = Fconv.data[i].imag;
//        //yob[i] = Fconv.data[i];
//        //std::cout << yob[i] << std::endl;
//    }
//    yob.cp_to_device();
//    if(do_nag) {
//        for(int i = 0; i < img_size; i++){
//            img[i] = vol_out.data[i];
//        }
//        img.cp_to_device();
//    } else 
//        img.device_init(0.f);
//    transformer.fouriers.cp_to_device();
//    transformer.fouriers.streamSync();
//    transformer.backward(yob);
//    //img.cp_to_device();
//    yob.streamSync();
//    img.streamSync();
//    //yob.dump_device_to_file("yob"+devBundle->device_id);
//    RFLOAT yob_norm = getSquareSumOnBlock(yob);
//    yob_norm = sqrt(yob_norm/img_size);
//    //img.streamSync();
//    grads.device_alloc();
//    momentum.device_alloc();
//    momentum.device_init(0.f);
//    //move Fweight from host to device
//    weight.device_alloc();
//    for(int i = 0; i < Fweight.nzyxdim; i++){
//        weight[i] = Fweight.data[i];
//    }
//    weight.cp_to_device();
//    weight.streamSync();
//    RFLOAT tv_eps = 0.2;
//    if(normfft > 1. && data_dim == 2)
//        l_r /= normfft;
//    int FBsize = (int) ceilf((float)transformer.fouriers.getSize()/(float)BLOCK_SIZE);
//    int imgBsize = (int) ceilf((float)img.getSize()/(float)BLOCK_SIZE);
//    cuda_kernel_multi<<<imgBsize, BLOCK_SIZE, 0, yob.getStream()>>>(
//                ~yob,
//                (XFLOAT)1/((XFLOAT)normfft),
//                yob.getSize());
//    yob.streamSync();
//    RFLOAT weight_norm = getSquareSumOnBlock(weight);
//    weight_norm = sqrt(weight_norm/weight.getSize());
//    //std::cout << "device: " << devBundle->device_id << " Mout: " << weight_norm << std::endl;
//    l_r *= 1./(weight_norm);
//    std::cout << "start optimizing " << "yob: " << yob_norm << ", " << "weight: " << weight_norm << ", l_r: " << l_r << ", " << std::endl;
//    tv_alpha *= yob_norm;
//    tv_beta *= yob_norm;
//    std::cout << "tv_alpha: " << tv_alpha << ", tv_beta: " << tv_beta << std::endl;
//    for(int m_c = 0; m_c <= tv_iters; m_c++){
//        //forward transform img
//        img.cp_on_device(transformer.reals.d_ptr);
//        transformer.reals.streamSync();
//        transformer.forward();
//        transformer.fouriers.streamSync();
//        //multiply with weight and normalization factor
//        
//        cuda_kernel_complex_multi<<<FBsize, BLOCK_SIZE, 0, transformer.fouriers.getStream()>>>(
//                (XFLOAT*)~transformer.fouriers,
//                ~weight,
//                (XFLOAT)1/((XFLOAT)transformer.reals.getSize()),
//                transformer.fouriers.getSize());
//        transformer.fouriers.streamSync();
//        //inverse transform and put the transformation on grads
//        transformer.backward(grads);
//        grads.streamSync();
//        //transformer.backward();
//        //transformer.reals.cp_on_device(grads.d_ptr);
//        //substract Mout
//        cuda_kernel_substract<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
//                ~grads,
//                ~yob,
//                grads.getSize());
//        grads.streamSync();
//        //get the norm of gradient
//        //XFLOAT grads_norm = getSquareSumOnBlock(grads);
//        //grads_norm = sqrt(grads_norm);
//        //cuda_kernel_multi<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
//        //        ~grads,
//        //        (XFLOAT)1/((XFLOAT)grads_norm),
//        //        grads.getSize());
//        //grads.streamSync();
//        //get graph gradient
//        int Z = vol_out.zdim;
//        int Y = vol_out.ydim;
//        int X = vol_out.xdim;
//        if(data_dim == 3){
//            cuda_kernel_graph_grad<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
//                    ~img,
//                    ~grads,
//                    Z,
//                    Y,
//                    X,
//                    tv_beta,
//                    eps,//eps of log approximation
//                    tv_eps,//eps of l1 norm approximation
//                    grads.getSize());
//        } else {
//            cuda_kernel_graph_grad<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
//                    ~img,
//                    ~grads,
//                    Y,
//                    X,
//                    tv_beta,
//                    tv_eps,
//                    grads.getSize());
//
//        }
//        grads.streamSync();
//        //update momentum
//        //cuda_kernel_update_momentum<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
//        //        ~grads,
//        //        ~momentum,
//        //        mu,
//        //        l_r,
//        //        grads.getSize());
//        //grads.streamSync();
//        //update momentum and soft thresholding
//        cuda_kernel_soft_threshold<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
//                ~img,
//                ~momentum,
//                ~grads,
//                mu,
//                l_r,
//                tv_alpha,
//                eps,
//                grads.getSize());
//        img.streamSync();
//        if(m_c % 20 == 0){
//            std::cout << m_c << " ";
//            RFLOAT grads_norm = getSquareSumOnBlock(img);
//            RFLOAT resi_norm = getSquareSumOnBlock(grads);
//            grads_norm = sqrt(grads_norm/img_size);
//            resi_norm = sqrt(resi_norm/img_size);
//            std::cout <<  " " << resi_norm << " " << grads_norm << " " << resi_norm/grads_norm << std::endl;
//        }
//        l_r *= exp(-0.005);
//    }
//    //now copy image to host
//    img.cp_to_host();
//    //now set vol_out
//    bool hasnan = false;
//    for(int i = 0; i < img_size; i++){
//        if(isnan(img[i])){
//            vol_out.data[i] = 0.;
//            hasnan = true;
//        }
//        else
//            vol_out.data[i] = img[i];
//    }
//    if(hasnan) std::cout << "WARNING: find nan in reconstruction." << std::endl;
//    transformer.clear();
//    img.free();
//    yob.free();
//    weight.free();
//    grads.free();
//    momentum.free();
//    //hipDeviceReset();
//}
