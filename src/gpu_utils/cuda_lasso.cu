#include "hip/hip_runtime.h"
#include "src/gpu_utils/cuda_lasso.cuh"
#include <iomanip>
#include <signal.h>
#include <queue>
#include "src/fftw.h"
#include <math.h>
#include "src/gpu_utils/cuda_utils_cub.cuh"
#include "src/gpu_utils/cuda_fft.h"
#include "src/gpu_utils/cuda_kernels/helper.cuh"

inline int mapToCompact(int k, int i, int j, int Z, int Y, int X, int ZZ, int YY, int XX){
    if(k >= ZZ/2) k += ZZ;
    if(i >= YY/2) i += YY;
    if(j >= XX/2) j += XX;
    return k*Y*X + i*X + j;
}

void cuda_lasso(int fsc143, int tv_iters, RFLOAT l_r, RFLOAT mu, RFLOAT tv_alpha, RFLOAT tv_beta, MultidimArray<RFLOAT> &Fconv,
        MultidimArray<RFLOAT> &Fweight, MultidimArray<Complex> &Ftest_conv, MultidimArray<RFLOAT> &Ftest_weight, MultidimArray<RFLOAT> &vol_out, MlDeviceBundle *devBundle, int data_dim, RFLOAT normalise, RFLOAT nrparts, bool do_nag, RFLOAT implicit_weight, RFLOAT eps, RFLOAT epsp){
    hipSetDevice(devBundle->device_id);
    devBundle->setStream();
    std::cout <<" Device: " << devBundle->device_id <<", fsc143: " << fsc143;
    std::cout << " vol_out: " << vol_out.xdim << ", " << vol_out.ydim << ", " << vol_out.zdim << std::endl;
    std::cout << "Ftest_conv: " << Ftest_conv.xdim << ", " << Ftest_conv.ydim << ", " << Ftest_conv.zdim << std::endl;
    int ZZ = vol_out.zdim >> 1;
    int YY = vol_out.ydim >> 1;
    int XX = vol_out.xdim >> 1;
    int Z = vol_out.zdim;
    int Y = vol_out.ydim;
    int X = vol_out.xdim;

    int img_size = vol_out.nzyxdim;
    int img_size_h = 0;

    if(data_dim == 3) img_size_h = img_size/8;
    else img_size_h = img_size/4;

    std::priority_queue<XFLOAT, std::vector<XFLOAT>, std::greater<XFLOAT>> pq;
    CudaUnifedPtr<XFLOAT> img(devBundle->stream, devBundle->device_id, img_size);
    //momentum term
    //CudaUnifedPtr<XFLOAT> momentum(devBundle->stream, devBundle->device_id, img_size);
    CudaUnifedPtr<XFLOAT> yob(devBundle->stream, devBundle->device_id, img_size);
    CudaUnifedPtr<XFLOAT> grads(devBundle->stream, devBundle->device_id, img_size);
    CudaUnifedPtr<XFLOAT> sigma_norm(devBundle->stream, devBundle->device_id, 1);
    XFLOAT mem_size = vol_out.getSize()*sizeof(XFLOAT)/1024./1024.;
    //std::cout << "Mem size " << 4*mem_size << " " << 3*img_size_h*sizeof(XFLOAT)/1024./1024. << "MB "<< std::endl;
    //CudaFFTT<false> transformer(devBundle->stream, NULL, data_dim);
    CudaFFTU transformer(devBundle->stream, devBundle->device_id, data_dim);
    transformer.setSize(vol_out.xdim, vol_out.ydim, vol_out.zdim);
    CudaUnifedPtr<XFLOAT> weight(devBundle->stream, devBundle->device_id, Fweight.getSize());
    //img.device_alloc();
    img.setPtr(transformer.reals.ptr);
    //img.alloc();
    //momentum is used to calculate grads
    //momentum.setPtr(transformer.reals.ptr);
    yob.alloc();
    sigma_norm.alloc();
    XFLOAT lambda = implicit_weight*normalise;
    //std::cout << "weight " << weight.getSize() << ", " << Fconv.getSize()<< std::endl;
    RFLOAT fconv_norm = 0.;
    XFLOAT yob_norm = 0.;
    int median_size = img_size_h/16;
    int sparse_count = 0;

    if(do_nag) {
        for(int i = 0; i < Fconv.nzyxdim; i++){
            //img[i] = 0.f;
            //transformer.fouriers[i].x = Fconv.data[i].real;
            //transformer.fouriers[i].y = Fconv.data[i].imag;
            yob[i] = Fconv.data[i];
            fconv_norm += Fconv.data[i]*Fconv.data[i];
            if(pq.size() < median_size) pq.push(yob[i]);
            else if(pq.top() < yob[i]) {
                pq.pop();
                pq.push(yob[i]);
            }
            yob[i] += lambda*((XFLOAT)vol_out.data[i]);
            //std::cout << yob[i] << std::endl;
        }
        fconv_norm = sqrt(fconv_norm/Fconv.nzyxdim);
        //for(int k = 0; k < ZZ; k++)
        //    for(int i = 0; i < YY; i++)
        //        for(int j = 0; j < XX; j++){
        //            int index = mapToCompact(k, i, j, Z, Y, X, ZZ, YY, XX);
        //            int my_index = k*YY*XX + i*XX + j;
        //            yob[my_index] = Fconv.data[index];
        //            yob[my_index] += lambda*((XFLOAT)vol_out.data[index]);
        //        }
        //for(int i = 0; i < img_size; i++){
        //    img[i] = vol_out.data[i];
        //    //momentum[i] = momentum_out.data[i];
        //}
        //momentum.device_init(0.f);
        //vol_out is of even size
        FOR_ALL_DIRECT_ELEMENTS_IN_ARRAY3D(vol_out){
            if((k < ZZ || k >= Z - ZZ) &&
               (i < YY || i >= Y - YY) &&
               (j < XX || j >= X - XX)){
                img[k*Y*X + i*X + j] = DIRECT_A3D_ELEM(vol_out, k, i, j);
                if(img[k*Y*X + i*X + j] > 1.e-3) {
                    sparse_count += 1;
                    yob_norm += abs(img[k*Y*X + i*X + j]);
                } else {
                    //img[k*Y*X + i*X + j] = 0.;
                }
            } else {
                img[k*Y*X + i*X + j] = 0.;
            }
            //momentum[k*Y*X + i*X + j] = img[k*Y*X + i*X + j];
        }
        //img.init();
    } else {
        for(int i = 0; i < Fconv.nzyxdim; i++){
            //img[i] = 0.f;
            //transformer.fouriers[i].x = Fconv.data[i].real;
            //transformer.fouriers[i].y = Fconv.data[i].imag;
            yob[i] = Fconv.data[i];
            //std::cout << yob[i] << std::endl;
        }
        img.init();
        //momentum.device_init(0.f);
    }

    //adjust median according to sparseness count
    //if(sparse_count*0.5 < median_size){
    //    int new_median_size = sparse_count*0.5;
    //    for(int i = 0; i < median_size - new_median_size; i++){
    //        pq.pop();
    //    }
    //}
    std::cout << "median: " << pq.top() << " pq_fraction: " << pq.size()/XFLOAT(img_size_h) << " sparseness: " << sparse_count/XFLOAT(img_size_h) << std::endl;
    //yob.cp_to_device();
    yob.set_read_only();
    yob.attach_to_stream();
    img.attach_to_stream();
    transformer.fouriers.attach_to_device();
    transformer.fouriers.attach_to_stream();
    //yob.attach_to_device();
    //transformer.fouriers.cp_to_device();
    //transformer.backward(yob);
    //img.cp_to_device();
    //yob.dump_device_to_file("yob"+devBundle->device_id);
    
    grads.alloc();
    grads.attach_to_stream();
    //move Fweight from host to device
    //size_t free_byte ;
    //size_t total_byte ;
    //hipMemGetInfo( &free_byte, &total_byte ) ;
    //std::cout << "Free Mem " << free_byte/1024./1024. << " Total Mem " << total_byte/1024./1024. << std::endl;

    weight.alloc();
    //int xdim = Fconv.xdim/2 + 1;
    XFLOAT max_weight = 0;
    XFLOAT min_weight = normalise;
    FOR_ALL_ELEMENTS_IN_FFTW_TRANSFORM(Fweight){
        int kw = kp;
        int iw = ip;
        int jw = jp;
        if(kp < 0) kw += Fweight.zdim;
        if(ip < 0) iw += Fweight.ydim;
        int index = kw*Fweight.ydim*Fweight.xdim + iw*Fweight.xdim + jw;
        weight[index] = FFTW_ELEM(Fweight, kp, ip, jp);
        if(max_weight < weight[index]) max_weight = weight[index];
        if(min_weight > weight[index] && weight[index] > 1e-5) min_weight = weight[index];        
    }
    weight.set_read_only();
    weight.attach_to_stream();
    //synchronize before launch kernel
    img.streamSync();
    weight.cp_to_device();
    yob_norm /= img_size_h;
    yob_norm = sqrt(yob_norm);
    //eps = yob_norm*1.;
    //eps = max(pq.top(), 0.1);
    //eps = 0.1;
    XFLOAT tv_eps = 1./sqrt(normalise);//0.00005;
    XFLOAT tv_log_eps = eps;
    int FBsize = (int) ceilf((float)transformer.fouriers.getSize()/(float)BLOCK_SIZE);
    int imgBsize = (int) ceilf((float)img_size_h/(float)BLOCK_SIZE);
    int imgBFsize = (int) ceilf((float)img.getSize()/(float)BLOCK_SIZE);
    XFLOAT weight_norm = getSquareSumOnBlock(weight);
    weight_norm = sqrt(weight_norm/weight.getSize());
    
    std::cout << "device: " << devBundle->device_id << " weight: " << weight_norm << std::endl;
    Fweight.printShape();
    l_r = l_r/(max_weight + normalise);
    std::cout << "start optimizing " << "lambda: " << lambda << " avg weight : " << normalise << " max weight: " << max_weight << " min weight: " << min_weight << " condition number: " << max_weight/min_weight << std::endl;
    //tv_alpha *= std::sqrt(normalise);
    //tv_beta *= std::sqrt(normalise);
    std::cout << "fconv_norm: " << fconv_norm << " img_norm: " << yob_norm <<" img_size " << img_size << " FBsize " << FBsize << std::endl;
    XFLOAT scale = (XFLOAT)1/((XFLOAT)transformer.reals.getSize());
    //XFLOAT w = tv_alpha/4;///eps*0.025;
    XFLOAT w = sqrt(normalise)/3;///(X*3);
    //optimize parameter here
    //set initial value for every parameter set
    //for storing best result
    MultidimArray<RFLOAT> best_img(vol_out, true);
    RFLOAT min_err = img_size_h;
    RFLOAT alpha = tv_alpha;
    RFLOAT beta = tv_beta;
    RFLOAT best_beta = tv_beta;
    RFLOAT best_eps = eps;
    for(int eps_i = 0; eps_i < 2; eps_i++){
        //eps = 0.05/(eps_i+1);
        //eps = 0.01;
        tv_log_eps = epsp;//eps*2;
        tv_alpha = alpha/(eps_i+1.)*fconv_norm*eps;//sqrt(normalise);//fconv_norm*eps/3;
        for(int beta_i = 0; beta_i < 5; beta_i++){
            tv_beta = beta/(eps_i+1.)*(1. - float(beta_i)/5.)*fconv_norm*tv_log_eps;//sqrt(normalise);//fconv_norm*eps/3;
            w = tv_alpha;
            for(int m_c = 0; m_c <= tv_iters; m_c++){
                //forward transform img/momentum
                transformer.forward();
                //multiply with weight and normalization factor
                //cuda_kernel_complex_multi<<<FBsize, BLOCK_SIZE, 0, transformer.fouriers.getStream()>>>(
                //        (XFLOAT*)~transformer.fouriers,
                //        ~weight,
                //        (XFLOAT)1/((XFLOAT)transformer.reals.getSize()),
                //        transformer.fouriers.getSize());
                int xdim = X/2 + 1;
                cuda_kernel_complex_multi<<<FBsize, BLOCK_SIZE, 0, transformer.fouriers.getStream()>>>(
                        (XFLOAT*)~transformer.fouriers,
                        ~weight,
                        scale,
                        w*scale,
                        Z,
                        Y,
                        xdim,
                        Fweight.zdim,
                        Fweight.ydim,
                        Fweight.xdim,
                        transformer.fouriers.getSize());
                //inverse transform and put the transformation on grads
                transformer.backward(grads);
                //transformer.reals.cp_on_device(grads.d_ptr);
                //substract Mout
                //cuda_kernel_substract<<<imgBFsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                //        ~grads,
                //        ~yob,
                //        grads.getSize());
                //work on windowed grads
                cuda_kernel_substract<<<imgBFsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                        ~grads,
                        ~yob,
                        ~img,
                        //~momentum,
                        //~vol,
                        (XFLOAT)lambda,
                        //ZZ,
                        //YY,
                        //XX,
                        //img_size_h);
                        grads.getSize());
                //synchronize before returning sum
                //get graph gradient
                if(data_dim == 3){
                    //work on windowed grads
                    cuda_kernel_graph_grad<<<imgBFsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                            ~img,
                            //~momentum,
                            ~grads,
                            Z,
                            Y,
                            X,
                            //ZZ,
                            //YY,
                            //XX,
                            tv_beta,
                            tv_log_eps,//eps of log approximation
                            tv_eps,//eps of l1 norm approximation
                            grads.getSize());
                    //img_size_h);
                } else {
                    cuda_kernel_graph_grad<<<imgBFsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                            ~img,
                            ~grads,
                            Y,
                            X,
                            tv_beta,
                            tv_eps,
                            grads.getSize());

                }
                cuda_kernel_soft_threshold<<<imgBFsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                        ~img,
                        //~momentum,
                        ~grads,
                        //mu,
                        l_r,
                        tv_alpha,
                        eps,
                        //X,
                        //Y,
                        //Z,
                        //XX,
                        //YY,
                        //ZZ,
                        grads.getSize());
                //img_size_h);
                //if(m_c && m_c % 100 == 0){
                //    std::cout << m_c << " ";
                //    RFLOAT img_norm = getSquareSumOnBlock(img);
                //    RFLOAT resi_norm = getSquareSumOnBlock(grads);
                //    img_norm = sqrt(img_norm/img_size_h);
                //    resi_norm = sqrt(resi_norm/img_size_h);
                //    std::cout << "beta: " << tv_beta << " " << resi_norm << " " << img_norm << " " << resi_norm/img_norm << std::endl;
                //}
                //l_r *= exp(-0.0025);
            }
            //get fourier transform of current model
            transformer.forward();
            transformer.fouriers.streamSync();
            //sync then
            //compare with test data
            RFLOAT test_err = 0.;
            RFLOAT test_counter = 0.;
            RFLOAT avg_F = 0.;
            FOR_ALL_ELEMENTS_IN_FFTW_TRANSFORM(Ftest_weight){
                if(kp*kp + ip*ip + jp*jp <= 4*fsc143*fsc143) {
                    int kw = kp;
                    int iw = ip;
                    int jw = jp;
                    if(kp < 0) kw += Ftest_conv.zdim;
                    if(ip < 0) iw += Ftest_conv.ydim;
                    int index = kw*Ftest_conv.ydim*Ftest_conv.xdim + iw*Ftest_conv.xdim + jw;
                    RFLOAT w = FFTW_ELEM(Ftest_weight, kp, ip, jp);
                    if (w < 1.) continue;
                    RFLOAT diff_x = transformer.fouriers[index].x*scale - Ftest_conv.data[index].real/w;
                    RFLOAT diff_y = transformer.fouriers[index].y*scale - Ftest_conv.data[index].imag/w;
                    test_err += (diff_x*diff_x + diff_y*diff_y)*w;
                    avg_F += (Ftest_conv.data[index].real/w)*(Ftest_conv.data[index].real/w)*w;
                    avg_F += (Ftest_conv.data[index].imag/w)*(Ftest_conv.data[index].imag/w)*w;
                    test_counter += 1;
                }
            }
            test_err /= test_counter;
            avg_F /= test_counter;
            std::cout << "eps_i: " << eps_i << " " << beta_i << " beta_i: " << tv_beta << " test_err: " << sqrt(test_err) << " avg_F: " << sqrt(avg_F) << std::endl;
            if(min_err > test_err) {
                min_err = test_err;
                best_beta = tv_beta;
                best_eps = eps;
                //store result in best_img
                for(int i = 0; i < img_size; i++){
                    best_img.data[i] = img[i];//(1. + w/img_size_h)*img[i];
                }
            }
        }//end beta search loop
    }//end eps search loop
    //print best beta
    std::cout << "eps: " << best_eps << " tv_beta: " << best_beta << " min_err: " << sqrt(min_err) << std::endl;
    //now copy image to host
    //now set vol_out and wait for stream to complete
    img.streamSync();
    bool hasnan = false;
    for(int i = 0; i < img_size; i++){
        if(isnan(best_img.data[i])){
            vol_out.data[i] = 0.;
            hasnan = true;
        }
        else
        {
            //consider upscale image
            vol_out.data[i] = best_img.data[i];
        }
    }
    if(hasnan) std::cout << "WARNING: find nan in reconstruction." << std::endl;
    transformer.clear();
    devBundle->destroyStream();
}

