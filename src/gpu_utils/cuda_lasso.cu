#include "hip/hip_runtime.h"
#include "src/gpu_utils/cuda_lasso.cuh"
#include <signal.h>
#include <math.h>
#include "src/gpu_utils/cuda_utils_cub.cuh"
#include "src/gpu_utils/cuda_fft.h"
#include "src/gpu_utils/cuda_kernels/helper.cuh"
#include <stack>

void cuda_lasso(int tv_iters, RFLOAT l_r, RFLOAT mu, RFLOAT tv_alpha, RFLOAT tv_beta, RFLOAT eps, MultidimArray<RFLOAT> &Fconv,
        MultidimArray<RFLOAT> &Fweight, MultidimArray<RFLOAT> &vol_out, MlDeviceBundle *devBundle, int data_dim, RFLOAT normfft){
    //normfft = max(normfft, 1.);
    hipSetDevice(devBundle->device_id);
    int img_size = vol_out.nzyxdim;
    CudaGlobalPtr<XFLOAT, false> img(img_size, devBundle->stream);
    CudaGlobalPtr<XFLOAT, false> yob(img_size, devBundle->stream);
    CudaGlobalPtr<XFLOAT, false> weight(Fweight.nzyxdim, devBundle->stream);
    CudaGlobalPtr<XFLOAT, false> grads(img_size, devBundle->stream);
    CudaGlobalPtr<XFLOAT, false> momentum(img_size, devBundle->stream);
    CudaFFTT<false> transformer(devBundle->stream, 0, data_dim);
    transformer.setSize(vol_out.xdim, vol_out.ydim, vol_out.zdim);
    img.device_alloc();
    yob.device_alloc();
    for(int i = 0; i < Fconv.getSize(); i++){
        //img[i] = 0.f;
        //transformer.fouriers[i].x = Fconv.data[i].real;
        //transformer.fouriers[i].y = Fconv.data[i].imag;
        yob[i] = Fconv.data[i];
    }
    //transformer.fouriers.cp_to_device();
    //transformer.fouriers.streamSync();
    //transformer.backward(yob);
    //img.cp_to_device();
    yob.cp_to_device();
    yob.streamSync();
    img.device_init(0.f);
    //img.streamSync();
    grads.device_alloc();
    momentum.device_alloc();
    momentum.device_init(0.f);
    //move Fweight from host to device
    weight.device_alloc();
    for(int i = 0; i < Fweight.nzyxdim; i++){
        weight[i] = Fweight.data[i];
    }
    weight.cp_to_device();
    weight.streamSync();
    RFLOAT tv_eps = 0.2;
    if(normfft > 1.)
        l_r /= normfft;
    int FBsize = (int) ceilf((float)transformer.fouriers.getSize()/(float)BLOCK_SIZE);
    int imgBsize = (int) ceilf((float)img.getSize()/(float)BLOCK_SIZE);
    //cuda_kernel_multi<<<imgBsize, BLOCK_SIZE, 0, yob.getStream()>>>(
    //            ~yob,
    //            (XFLOAT)1/((XFLOAT)normfft),
    //            yob.getSize());
    //yob.streamSync();
    //XFLOAT weight_norm = getSquareSumOnDevice(yob);
    //weight_norm = sqrt(weight_norm/img_size);
    //std::cout << "device: " << devBundle->device_id << " Mout: " << weight_norm << std::endl;
    std::cout << "start optimizing " << l_r << ", " << weight.getSize() << std::endl;

    for(int m_c = 0; m_c <= tv_iters; m_c++){
        //forward transform img
        img.cp_on_device(transformer.reals.d_ptr);
        transformer.reals.streamSync();
        transformer.forward();
        transformer.fouriers.streamSync();
        //multiply with weight and normalization factor
        
        cuda_kernel_complex_multi<<<FBsize, BLOCK_SIZE, 0, transformer.fouriers.getStream()>>>(
                (XFLOAT*)~transformer.fouriers,
                ~weight,
                (XFLOAT)1/((XFLOAT)transformer.reals.getSize()),
                transformer.fouriers.getSize());
        transformer.fouriers.streamSync();
        //inverse transform and put the transformation on grads
        transformer.backward(grads);
        //transformer.backward();
        //transformer.reals.cp_on_device(grads.d_ptr);
        //substract Mout
        cuda_kernel_substract<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                ~grads,
                ~yob,
                grads.getSize());
        grads.streamSync();
        //get the norm of gradient
        //XFLOAT grads_norm = getSquareSumOnBlock(grads);
        //grads_norm = sqrt(grads_norm);
        //cuda_kernel_multi<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
        //        ~grads,
        //        (XFLOAT)1/((XFLOAT)normfft),
        //        grads.getSize());
        //grads.streamSync();
        //get graph gradient
        int Z = vol_out.zdim;
        int Y = vol_out.ydim;
        int X = vol_out.xdim;
        if(data_dim == 3){
            cuda_kernel_graph_grad<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                    ~img,
                    ~grads,
                    Z,
                    Y,
                    X,
                    tv_beta,
                    eps,//eps of log approximation
                    tv_eps,//eps of l1 norm approximation
                    grads.getSize());
        } else {
            cuda_kernel_graph_grad<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                    ~img,
                    ~grads,
                    Y,
                    X,
                    tv_beta,
                    tv_eps,
                    grads.getSize());

        }
        grads.streamSync();
        //update momentum
        //cuda_kernel_update_momentum<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
        //        ~grads,
        //        ~momentum,
        //        mu,
        //        l_r,
        //        grads.getSize());
        //grads.streamSync();
        //update momentum and soft thresholding
        cuda_kernel_soft_threshold<<<imgBsize, BLOCK_SIZE, 0, grads.getStream()>>>(
                ~img,
                ~momentum,
                ~grads,
                mu,
                l_r,
                tv_alpha,
                eps,
                grads.getSize());
        img.streamSync();
        if(m_c % 100 == 0){
            //std::cout << m_c << " " << grads_norm;
            RFLOAT grads_norm = getSquareSumOnBlock(img);
            RFLOAT resi_norm = getSquareSumOnBlock(grads);
            grads_norm = sqrt(grads_norm/img_size);
            resi_norm = sqrt(resi_norm/img_size);
            std::cout <<  m_c << " " << grads_norm << " " << resi_norm/grads_norm << std::endl;
        }
        l_r *= exp(-0.005);
    }
    //now copy image to host
    img.cp_to_host();
    //now set vol_out
    bool hasnan = false;
    for(int i = 0; i < img_size; i++){
        if(isnan(img[i])){
            vol_out.data[i] = 0.;
            hasnan = true;
        }
        else
            vol_out.data[i] = img[i];
    }
    if(hasnan) std::cout << "WARNING: find nan in reconstruction." << std::endl;
    transformer.clear();
    img.free();
    yob.free();
    weight.free();
    grads.free();
    momentum.free();
    //hipDeviceReset();
}
