#include "hip/hip_runtime.h"
#include "src/gpu_utils/cuda_device_utils.cuh"
#include "src/gpu_utils/cuda_kernels/helper.cuh"
#include "src/gpu_utils/cuda_settings.h"

/*
 * This draft of a kernel assumes input that has jobs which have a single orientation and sequential translations within each job.
 *
 */
__global__ void cuda_kernel_exponentiate_weights_fine(
		XFLOAT *g_pdf_orientation,
		XFLOAT *g_pdf_offset,
		XFLOAT *g_weights,
		XFLOAT avg_diff2,
		int oversamples_orient,
		int oversamples_trans,
		unsigned long *d_rot_id,
		unsigned long *d_trans_idx,
		unsigned long *d_job_idx,
		unsigned long *d_job_num,
		long int job_num)
{
	__shared__ XFLOAT s_weights[SUMW_BLOCK_SIZE];

	// blockid
	int bid  = blockIdx.x;
	//threadid
	int tid = threadIdx.x;

	long int jobid = bid*SUMW_BLOCK_SIZE+tid;

	if (jobid<job_num)
	{
		long int pos = d_job_idx[jobid];
		// index of comparison
		long int ix =  d_rot_id[   pos];   // each thread gets its own orient...
		long int iy = d_trans_idx[ pos];   // ...and it's starting trans...
		long int in =  d_job_num[jobid];    // ...AND the number of translations to go through

		int c_itrans;//, iorient = bid*SUM_BLOCK_SIZE+tid; //, f_itrans;

		// Bacause the partion of work is so arbitrarily divided in this kernel,
		// we need to do some brute idex work to get the correct indices.
		for (int itrans=0; itrans < in; itrans++, iy++)
		{
			c_itrans = ( iy - (iy % oversamples_trans))/ oversamples_trans; //floor(x/y) == (x-(x%y))/y  but less sensitive to x>>y and finite precision
//			f_itrans = iy % oversamples_trans;

			XFLOAT prior = g_pdf_orientation[ix] * g_pdf_offset[c_itrans];          	// Same      for all threads - TODO: should be done once for all trans through warp-parallel execution
			XFLOAT diff2 = g_weights[pos+itrans] - avg_diff2;								// Different for all threads
			// next line because of numerical precision of exp-function
	#if defined(CUDA_DOUBLE_PRECISION)
				if (diff2 > 700.)
					s_weights[tid] = 0.;
				else
					s_weights[tid] = prior * exp(-diff2);
	#else
				if (diff2 > 86.)
					s_weights[tid] = 0.f;
				else
					s_weights[tid] = prior * expf(-diff2);
	#endif
				// TODO: use tabulated exp function? / Sjors  TODO: exp, expf, or __exp in CUDA? /Bjorn
			// Store the weight
			g_weights[pos+itrans] = s_weights[tid]; // TODO put in shared mem
		}
	}
}

__global__ void cuda_kernel_softMaskOutsideMap(	XFLOAT *vol,
												long int vol_size,
												long int xdim,
												long int ydim,
												long int zdim,
												long int xinit,
												long int yinit,
												long int zinit,
												bool do_Mnoise,
												XFLOAT radius,
												XFLOAT radius_p,
												XFLOAT cosine_width	)
{

		int tid = threadIdx.x;

//		vol.setXmippOrigin(); // sets xinit=xdim , also for y z
		XFLOAT r, raisedcos;

		__shared__ XFLOAT     img_pixels[SOFTMASK_BLOCK_SIZE];
		__shared__ XFLOAT    partial_sum[SOFTMASK_BLOCK_SIZE];
		__shared__ XFLOAT partial_sum_bg[SOFTMASK_BLOCK_SIZE];

		XFLOAT sum_bg_total =  (XFLOAT)0.0;

		long int texel_pass_num = ceilfracf(vol_size,SOFTMASK_BLOCK_SIZE);
		int texel = tid;

		partial_sum[tid]=(XFLOAT)0.0;
		partial_sum_bg[tid]=(XFLOAT)0.0;
		if (do_Mnoise)
		{
			for (int pass = 0; pass < texel_pass_num; pass++, texel+=SOFTMASK_BLOCK_SIZE) // loop the available warps enough to complete all translations for this orientation
			{
				XFLOAT x,y,z;
				if(texel<vol_size)
				{
					img_pixels[tid]=__ldg(&vol[texel]);

					z = floor( (float) texel                   / (float)((xdim)*(ydim)));
					y = floor( (XFLOAT)(texel-z*(xdim)*(ydim)) / (XFLOAT) xdim );
					x = texel - z*(xdim)*(ydim) - y*xdim;

					z-=zinit;
					y-=yinit;
					x-=xinit;

					r = sqrt(x*x + y*y + z*z);

					if (r < radius)
						continue;
					else if (r > radius_p)
					{
						partial_sum[tid]    += (XFLOAT)1.0;
						partial_sum_bg[tid] += img_pixels[tid];
					}
					else
					{
#if defined(CUDA_DOUBLE_PRECISION)
						raisedcos = 0.5 + 0.5  * cospi( (radius_p - r) / cosine_width );
#else
						raisedcos = 0.5f + 0.5f * cospif((radius_p - r) / cosine_width );
#endif
						partial_sum[tid] += raisedcos;
						partial_sum_bg[tid] += raisedcos * img_pixels[tid];
					}
				}
			}
		}

		__syncthreads();
		for(int j=(SOFTMASK_BLOCK_SIZE/2); j>0; j/=2)
		{
			if(tid<j)
			{
				partial_sum[tid] += partial_sum[tid+j];
				partial_sum_bg[tid] += partial_sum_bg[tid+j];
			}
			__syncthreads();
		}

		sum_bg_total  = partial_sum_bg[0] / partial_sum[0];


		texel = tid;
		for (int pass = 0; pass < texel_pass_num; pass++, texel+=SOFTMASK_BLOCK_SIZE) // loop the available warps enough to complete all translations for this orientation
		{
			XFLOAT x,y,z;
			if(texel<vol_size)
			{
				img_pixels[tid]=__ldg(&vol[texel]);

				z =  floor( (float) texel                  / (float)((xdim)*(ydim)));
				y = floor( (XFLOAT)(texel-z*(xdim)*(ydim)) / (XFLOAT)  xdim         );
				x = texel - z*(xdim)*(ydim) - y*xdim;

				z-=zinit;
				y-=yinit;
				x-=xinit;

				r = sqrt(x*x + y*y + z*z);

				if (r < radius)
					continue;
				else if (r > radius_p)
					img_pixels[tid]=sum_bg_total;
				else
				{
#if defined(CUDA_DOUBLE_PRECISION)
					raisedcos = 0.5  + 0.5  * cospi( (radius_p - r) / cosine_width );
#else
					raisedcos = 0.5f + 0.5f * cospif((radius_p - r) / cosine_width );
#endif
					img_pixels[tid]= img_pixels[tid]*(1-raisedcos) + sum_bg_total*raisedcos;

				}
				vol[texel]=img_pixels[tid];
			}

		}
}

__global__ void cuda_kernel_softMaskBackgroundValue(	XFLOAT *vol,
														long int vol_size,
														long int xdim,
														long int ydim,
														long int zdim,
														long int xinit,
														long int yinit,
														long int zinit,
														bool do_Mnoise,
														XFLOAT radius,
														XFLOAT radius_p,
														XFLOAT cosine_width,
														XFLOAT *g_sum,
														XFLOAT *g_sum_bg)
{

		int tid = threadIdx.x;
		int bid = blockIdx.x;

//		vol.setXmippOrigin(); // sets xinit=xdim , also for y z
		XFLOAT r, raisedcos;
		int x,y,z;
		__shared__ XFLOAT     img_pixels[SOFTMASK_BLOCK_SIZE];
		__shared__ XFLOAT    partial_sum[SOFTMASK_BLOCK_SIZE];
		__shared__ XFLOAT partial_sum_bg[SOFTMASK_BLOCK_SIZE];

		long int texel_pass_num = ceilfracf(vol_size,SOFTMASK_BLOCK_SIZE*gridDim.x);
		int texel = bid*SOFTMASK_BLOCK_SIZE*texel_pass_num + tid;

		partial_sum[tid]=(XFLOAT)0.0;
		partial_sum_bg[tid]=(XFLOAT)0.0;

		for (int pass = 0; pass < texel_pass_num; pass++, texel+=SOFTMASK_BLOCK_SIZE) // loop the available warps enough to complete all translations for this orientation
		{
			if(texel<vol_size)
			{
				img_pixels[tid]=__ldg(&vol[texel]);

				z =   texel / (xdim*ydim) ;
				y = ( texel % (xdim*ydim) ) / xdim ;
				x = ( texel % (xdim*ydim) ) % xdim ;

				z-=zinit;
				y-=yinit;
				x-=xinit;

				r = sqrt(XFLOAT(x*x + y*y + z*z));

				if (r < radius)
					continue;
				else if (r > radius_p)
				{
					partial_sum[tid]    += (XFLOAT)1.0;
					partial_sum_bg[tid] += img_pixels[tid];
				}
				else
				{
#if defined(CUDA_DOUBLE_PRECISION)
					raisedcos = 0.5 + 0.5  * cospi( (radius_p - r) / cosine_width );
#else
					raisedcos = 0.5f + 0.5f * cospif((radius_p - r) / cosine_width );
#endif
					partial_sum[tid] += raisedcos;
					partial_sum_bg[tid] += raisedcos * img_pixels[tid];
				}
			}
		}

		cuda_atomic_add(&g_sum[tid]   , partial_sum[tid]);
		cuda_atomic_add(&g_sum_bg[tid], partial_sum_bg[tid]);
}


__global__ void cuda_kernel_cosineFilter(	XFLOAT *vol,
											long int vol_size,
											long int xdim,
											long int ydim,
											long int zdim,
											long int xinit,
											long int yinit,
											long int zinit,
											bool do_Mnoise,
											XFLOAT radius,
											XFLOAT radius_p,
											XFLOAT cosine_width,
											XFLOAT bg_value)
{

	int tid = threadIdx.x;
	int bid = blockIdx.x;

//		vol.setXmippOrigin(); // sets xinit=xdim , also for y z
	XFLOAT r, raisedcos;
	int x,y,z;
	__shared__ XFLOAT     img_pixels[SOFTMASK_BLOCK_SIZE];

	long int texel_pass_num = ceilfracf(vol_size,SOFTMASK_BLOCK_SIZE*gridDim.x);
	int texel = bid*SOFTMASK_BLOCK_SIZE*texel_pass_num + tid;

	for (int pass = 0; pass < texel_pass_num; pass++, texel+=SOFTMASK_BLOCK_SIZE) // loop the available warps enough to complete all translations for this orientation
	{
		if(texel<vol_size)
		{
			img_pixels[tid]=__ldg(&vol[texel]);

			z =   texel / (xdim*ydim) ;
			y = ( texel % (xdim*ydim) ) / xdim ;
			x = ( texel % (xdim*ydim) ) % xdim ;

			z-=zinit;
			y-=yinit;
			x-=xinit;

			r = sqrt(XFLOAT(x*x + y*y + z*z));

			if (r < radius)
				continue;
			else if (r > radius_p)
				img_pixels[tid]=bg_value;
			else
			{
#if defined(CUDA_DOUBLE_PRECISION)
				raisedcos = 0.5  + 0.5  * cospi( (radius_p - r) / cosine_width );
#else
				raisedcos = 0.5f + 0.5f * cospif((radius_p - r) / cosine_width );
#endif
				img_pixels[tid]= img_pixels[tid]*(1-raisedcos) + bg_value*raisedcos;

			}
			vol[texel]=img_pixels[tid];
		}

	}
}


__global__ void cuda_kernel_translate2D(	XFLOAT * g_image_in,
											XFLOAT * g_image_out,
											int image_size,
											int xdim,
											int ydim,
											int dx,
											int dy)
{
	int tid = threadIdx.x;
	int bid =  blockIdx.x;

	int x,y,xp,yp;
	int pixel=tid + bid*BLOCK_SIZE;
	int new_pixel;

	if(pixel<image_size)
	{
		x = pixel % xdim;
		y = (pixel-x) / (xdim);

		xp = x + dx;
		yp = y + dy;

		if( yp>=0 && xp>=0 && yp<ydim && xp<xdim)
		{
			new_pixel = yp*xdim + xp;
			if(new_pixel>=0 && new_pixel<image_size) // if displacement is negative, new_pixel could be less than 0
				g_image_out[new_pixel] = g_image_in[pixel];
		}
	}
}

__global__ void cuda_kernel_translate3D(	XFLOAT * g_image_in,
											XFLOAT * g_image_out,
											int image_size,
											int xdim,
											int ydim,
											int zdim,
											int dx,
											int dy,
											int dz)
{
	int tid = threadIdx.x;
	int bid =  blockIdx.x;

	int x,y,z,xp,yp,zp,xy;
	int voxel=tid + bid*BLOCK_SIZE;
	int new_voxel;

	int xydim = xdim*ydim;

	if(voxel<image_size)
	{
		z =  voxel / xydim;
		zp = z + dz;

		xy = voxel % xydim;
		y =  xy / xdim;
		yp = y + dy;

		x =  xy % xdim;
		xp = x + dx;

		if( zp>=0 && yp>=0 && xp>=0 && zp<zdim && yp<ydim && xp<xdim)
		{
			new_voxel = zp*xydim +  yp*xdim + xp;
			if(new_voxel>=0 && new_voxel<image_size) // if displacement is negative, new_pixel could be less than 0
				g_image_out[new_voxel] = g_image_in[voxel];
		}
	}
}

__global__ void cuda_kernel_centerFFT_2D(XFLOAT *img_in,
										 int image_size,
										 int xdim,
										 int ydim,
										 int xshift,
										 int yshift)
{

	__shared__ XFLOAT buffer[CFTT_BLOCK_SIZE];
	int tid = threadIdx.x;
	int pixel = threadIdx.x + blockIdx.x*CFTT_BLOCK_SIZE;
	long int image_offset = image_size*blockIdx.y;
//	int pixel_pass_num = ceilfracf(image_size, CFTT_BLOCK_SIZE);

//	for (int pass = 0; pass < pixel_pass_num; pass++, pixel+=CFTT_BLOCK_SIZE)
//	{
		if(pixel<(image_size/2))
		{
			int y = floorf((XFLOAT)pixel/(XFLOAT)xdim);
			int x = pixel % xdim;				// also = pixel - y*xdim, but this depends on y having been calculated, i.e. serial evaluation

			int yp = y + yshift;
			if (yp < 0)
				yp += ydim;
			else if (yp >= ydim)
				yp -= ydim;

			int xp = x + xshift;
			if (xp < 0)
				xp += xdim;
			else if (xp >= xdim)
				xp -= xdim;

			int n_pixel = yp*xdim + xp;

			buffer[tid]                    = img_in[image_offset + n_pixel];
			img_in[image_offset + n_pixel] = img_in[image_offset + pixel];
			img_in[image_offset + pixel]   = buffer[tid];
		}
//	}
}

__global__ void cuda_kernel_centerFFT_3D(XFLOAT *img_in,
										 int image_size,
										 int xdim,
										 int ydim,
										 int zdim,
										 int xshift,
										 int yshift,
									 	 int zshift)
{

	__shared__ XFLOAT buffer[CFTT_BLOCK_SIZE];
	int tid = threadIdx.x;
	int pixel = threadIdx.x + blockIdx.x*CFTT_BLOCK_SIZE;
	long int image_offset = image_size*blockIdx.y;

		int xydim = xdim*ydim;
		if(pixel<(image_size/2))
		{
			int z = floorf((XFLOAT)pixel/(XFLOAT)(xydim));
			int xy = pixel % xydim;
			int y = floorf((XFLOAT)xy/(XFLOAT)xdim);
			int x = xy % xdim;


			int yp = y + yshift;
			if (yp < 0)
				yp += ydim;
			else if (yp >= ydim)
				yp -= ydim;

			int xp = x + xshift;
			if (xp < 0)
				xp += xdim;
			else if (xp >= xdim)
				xp -= xdim;

			int zp = z + zshift;
			if (zp < 0)
				zp += zdim;
			else if (zp >= zdim)
				zp -= zdim;

			int n_pixel = zp*xydim + yp*xdim + xp;

			buffer[tid]                    = img_in[image_offset + n_pixel];
			img_in[image_offset + n_pixel] = img_in[image_offset + pixel];
			img_in[image_offset + pixel]   = buffer[tid];
		}
}


__global__ void cuda_kernel_probRatio(  XFLOAT *d_Mccf,
										XFLOAT *d_Mpsi,
										XFLOAT *d_Maux,
										XFLOAT *d_Mmean,
										XFLOAT *d_Mstddev,
										int image_size,
										XFLOAT normfft,
										XFLOAT sum_ref_under_circ_mask,
										XFLOAT sum_ref2_under_circ_mask,
										XFLOAT expected_Pratio,
										int NpsiThisBatch,
										int startPsi,
										int totalPsis)
{
	/* PLAN TO:
	 *
	 * 1) Pre-filter
	 * 		d_Mstddev[i] = 1 / (2*d_Mstddev[i])   ( if d_Mstddev[pixel] > 1E-10 )
	 * 		d_Mstddev[i] = 1    				  ( else )
	 *
	 * 2) Set
	 * 		sum_ref2_under_circ_mask /= 2.
	 *
	 * 3) Total expression becomes
	 * 		diff2 = ( exp(k) - 1.f ) / (expected_Pratio - 1.f)
	 * 	  where
	 * 	  	k = (normfft * d_Maux[pixel] + d_Mmean[pixel] * sum_ref_under_circ_mask)*d_Mstddev[i] + sum_ref2_under_circ_mask
	 *
	 */

	int pixel = threadIdx.x + blockIdx.x*(int)PROBRATIO_BLOCK_SIZE;
	if(pixel<image_size)
	{
		XFLOAT Kccf = d_Mccf[pixel];
		XFLOAT Kpsi =(XFLOAT)-1.0;
		for(int psi = 0; psi < NpsiThisBatch; psi++ )
		{
			XFLOAT diff2 = normfft * d_Maux[pixel + image_size*psi];
			diff2 += d_Mmean[pixel] * sum_ref_under_circ_mask;

	//		if (d_Mstddev[pixel] > (XFLOAT)1E-10)
			diff2 *= d_Mstddev[pixel];
			diff2 += sum_ref2_under_circ_mask;

#if defined(CUDA_DOUBLE_PRECISION)
			diff2 = exp(-diff2 / 2.); // exponentiate to reflect the Gaussian error model. sigma=1 after normalization, 0.4=1/sqrt(2pi)
#else
			diff2 = expf(-diff2 / 2.f);
#endif

			// Store fraction of (1 - probability-ratio) wrt  (1 - expected Pratio)
			diff2 = (diff2 - (XFLOAT)1.0) / (expected_Pratio - (XFLOAT)1.0);
			if (diff2 > Kccf)
			{
				Kccf = diff2;
				Kpsi = (startPsi + psi)*(360/totalPsis);
			}
		}
		d_Mccf[pixel] = Kccf;
		if (Kpsi >= 0.)
			d_Mpsi[pixel] = Kpsi;
	}
}

__global__ void cuda_kernel_rotateOnly(   CUDACOMPLEX *d_Faux,
						  	  	  	  	  XFLOAT psi,
						  	  			  CudaProjectorKernel projector,
						  	  			  int startPsi
						  	  			  )
{
	int proj = blockIdx.y;
	int image_size=projector.imgX*projector.imgY;
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
	{
		int y = floorfracf(pixel,projector.imgX);
		int x = pixel % projector.imgX;

		if (y > projector.maxR)
		{
			if (y >= projector.imgY - projector.maxR)
				y = y - projector.imgY;
			else
				x = projector.maxR;
		}

		XFLOAT sa, ca;
		sincos((proj+startPsi)*psi, &sa, &ca);
		CUDACOMPLEX val;

		projector.project2Dmodel(	 x,y,
									 ca,
									-sa,
									 sa,
									 ca,
									 val.x,val.y);

		long int out_pixel = proj*image_size + pixel;

		d_Faux[out_pixel].x =val.x;
		d_Faux[out_pixel].y =val.y;
	}
}

__global__ void cuda_kernel_rotateAndCtf( CUDACOMPLEX *d_Faux,
						  	  	  	  	  XFLOAT *d_ctf,
						  	  	  	  	  XFLOAT psi,
						  	  			  CudaProjectorKernel projector,
						  	  			  int startPsi
						  	  			  )
{
	int proj = blockIdx.y;
	int image_size=projector.imgX*projector.imgY;
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
	{
		int y = floorfracf(pixel,projector.imgX);
		int x = pixel % projector.imgX;

		if (y > projector.maxR)
		{
			if (y >= projector.imgY - projector.maxR)
				y = y - projector.imgY;
			else
				x = projector.maxR;
		}

		XFLOAT sa, ca;
		sincos((proj+startPsi)*psi, &sa, &ca);
		CUDACOMPLEX val;

		projector.project2Dmodel(	 x,y,
									 ca,
									-sa,
									 sa,
									 ca,
									 val.x,val.y);

		long int out_pixel = proj*image_size + pixel;

		d_Faux[out_pixel].x =val.x*d_ctf[pixel];
		d_Faux[out_pixel].y =val.y*d_ctf[pixel];

	}
}


__global__ void cuda_kernel_convol_A( CUDACOMPLEX *d_A,
									 CUDACOMPLEX *d_B,
									 int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
	{
		XFLOAT tr =   d_A[pixel].x;
		XFLOAT ti = - d_A[pixel].y;
		d_A[pixel].x =   tr*d_B[pixel].x - ti*d_B[pixel].y;
		d_A[pixel].y =   ti*d_B[pixel].x + tr*d_B[pixel].y;
	}
}

__global__ void cuda_kernel_convol_A( CUDACOMPLEX *d_A,
									 CUDACOMPLEX *d_B,
									 CUDACOMPLEX *d_C,
									 int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
	{
		XFLOAT tr =   d_A[pixel].x;
		XFLOAT ti = - d_A[pixel].y;
		d_C[pixel].x =   tr*d_B[pixel].x - ti*d_B[pixel].y;
		d_C[pixel].y =   ti*d_B[pixel].x + tr*d_B[pixel].y;
	}
}

__global__ void cuda_kernel_batch_convol_A( CUDACOMPLEX *d_A,
									 	 	CUDACOMPLEX *d_B,
									 	 	int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	int A_off = blockIdx.y*image_size;
	if(pixel<image_size)
	{
		XFLOAT tr =   d_A[pixel + A_off].x;
		XFLOAT ti = - d_A[pixel + A_off].y;
		d_A[pixel + A_off].x =   tr*d_B[pixel].x - ti*d_B[pixel].y;
		d_A[pixel + A_off].y =   ti*d_B[pixel].x + tr*d_B[pixel].y;
	}
}

__global__ void cuda_kernel_batch_convol_A( CUDACOMPLEX *d_A,
									 	 	CUDACOMPLEX *d_B,
									 	 	CUDACOMPLEX *d_C,
									 	 	int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	int A_off = blockIdx.y*image_size;
	if(pixel<image_size)
	{
		XFLOAT tr =   d_A[pixel + A_off].x;
		XFLOAT ti = - d_A[pixel + A_off].y;
		d_C[pixel + A_off].x =   tr*d_B[pixel].x - ti*d_B[pixel].y;
		d_C[pixel + A_off].y =   ti*d_B[pixel].x + tr*d_B[pixel].y;
	}
}

__global__ void cuda_kernel_convol_B(	 CUDACOMPLEX *d_A,
									 	 CUDACOMPLEX *d_B,
									 	 int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
	{
		XFLOAT tr = d_A[pixel].x;
		XFLOAT ti = d_A[pixel].y;
		d_A[pixel].x =   tr*d_B[pixel].x + ti*d_B[pixel].y;
		d_A[pixel].y =   ti*d_B[pixel].x - tr*d_B[pixel].y;
	}
}

__global__ void cuda_kernel_convol_B(	 CUDACOMPLEX *d_A,
									 	 CUDACOMPLEX *d_B,
									 	 CUDACOMPLEX *d_C,
									 	 int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
	{
		XFLOAT tr = d_A[pixel].x;
		XFLOAT ti = d_A[pixel].y;
		d_C[pixel].x =   tr*d_B[pixel].x + ti*d_B[pixel].y;
		d_C[pixel].y =   ti*d_B[pixel].x - tr*d_B[pixel].y;
	}
}

__global__ void cuda_kernel_batch_convol_B(	 CUDACOMPLEX *d_A,
									 	 	 CUDACOMPLEX *d_B,
									 	 	 int image_size)
{
	long int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	int A_off = blockIdx.y*image_size;
	if(pixel<image_size)
	{
		XFLOAT tr = d_A[pixel + A_off].x;
		XFLOAT ti = d_A[pixel + A_off].y;
		d_A[pixel + A_off].x =   tr*d_B[pixel].x + ti*d_B[pixel].y;
		d_A[pixel + A_off].y =   ti*d_B[pixel].x - tr*d_B[pixel].y;
	}
}

__global__ void cuda_kernel_multi( XFLOAT *A,
								   XFLOAT *OUT,
								   XFLOAT S,
		  	  	  	  	  	  	   int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
		OUT[pixel] = A[pixel]*S;
}

__global__ void cuda_kernel_multi(
		XFLOAT *A,
		XFLOAT S,
		int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
		A[pixel] = A[pixel]*S;
}

__global__ void cuda_kernel_multi( XFLOAT *A,
								   XFLOAT *B,
								   XFLOAT *OUT,
								   XFLOAT S,
		  	  	  	  	  	  	   int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size){
		OUT[pixel] = A[pixel]*B[pixel]*S;
    }
}

__global__ void cuda_kernel_complex_multi( XFLOAT *A,
                                   XFLOAT *B,
                                   XFLOAT S,
                                   int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size) {
        A[pixel*2] *= B[pixel]*S;
        A[pixel*2+1] *= B[pixel]*S;
    }
}

__global__ void cuda_kernel_complex_multi( XFLOAT *A,
                                   XFLOAT *B,
                                   XFLOAT S,
                                   XFLOAT w,
                                   int Z,
                                   int Y,
                                   int X,
                                   int ZZ,
                                   int YY,
                                   int XX,
                                   int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size) {
        int kp = pixel / (Y*X);
        int ip = (pixel - kp * (Y*X))/X;
        int jp = pixel - kp * (Y*X) - ip * X;
        if(kp >= X) kp -= (Z);
        if(ip >= X) ip -= (Y);
        if(kp < XX && kp > -XX && ip < XX && ip > -XX && jp < XX) {
            if(kp < 0) kp += ZZ;
            if(ip < 0) ip += YY;
            int n_pixel = kp*(YY*XX) + ip*XX + jp;
            A[pixel*2] *= (B[n_pixel]*S + w);
            A[pixel*2+1] *= (B[n_pixel]*S + w);
        } else {
            //A[pixel*2] = 0.;
            //A[pixel*2+1] = 0.;
            A[pixel*2] *=w;
            A[pixel*2+1] *=w;
        }
    }
}

__global__ void cuda_kernel_batch_multi( XFLOAT *A,
								   XFLOAT *B,
								   XFLOAT *OUT,
								   XFLOAT S,
		  	  	  	  	  	  	   int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
		OUT[pixel + blockIdx.y*image_size] = A[pixel + blockIdx.y*image_size]*B[pixel + blockIdx.y*image_size]*S;
}

__global__ void cuda_kernel_substract(XFLOAT *A,
                                     XFLOAT *B,
                                     int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size) {
        A[pixel] -= B[pixel];
    }
}

__global__ void cuda_kernel_substract(XFLOAT *A,
                                     XFLOAT *B,
                                     XFLOAT *C,
                                     XFLOAT l,
                                     int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size) {
        A[pixel] -= (B[pixel] - l*C[pixel]);
    }
}

__global__ void cuda_kernel_substract(XFLOAT *A,
                                     XFLOAT *B,
                                     XFLOAT *C,
                                     XFLOAT l,
                                     int Z,
                                     int Y,
                                     int X,
                                     int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size) {
        int kp = pixel / (Y*X);
        int ip = (pixel - kp * (Y*X))/X;
        int jp = pixel - kp * (Y*X) - ip * X;
        int hZ = Z >> 1;
        int hY = Y >> 1;
        int hX = X >> 1;
        if(kp >= hZ) kp += Z;
        if(ip >= hY) ip += Y;
        if(jp >= hX) jp += X;
        hY = Y << 1;
        hX = X << 1;
        int c_pixel = kp*hY*hX + ip*hX + jp;
        A[c_pixel] -= (B[c_pixel] - l*C[c_pixel]);
    }
}

__global__ void cuda_kernel_substract(XFLOAT *A,
                                     XFLOAT *B,
                                     XFLOAT *C,
                                     XFLOAT *vol_out,
                                     XFLOAT l,
                                     XFLOAT* sum,
                                     int Z,
                                     int Y,
                                     int X,
                                     int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size) {
        int kp = pixel / (Y*X);
        int ip = (pixel - kp * (Y*X))/X;
        int jp = pixel - kp * (Y*X) - ip * X;
        int hZ = Z >> 1;
        int hY = Y >> 1;
        int hX = X >> 1;
        if(kp >= hZ) kp += Z;
        if(ip >= hY) ip += Y;
        if(jp >= hX) jp += X;
        hY = Y << 1;
        hX = X << 1;
        int c_pixel = kp*hY*hX + ip*hX + jp;
        XFLOAT tmp = B[c_pixel] - vol_out[c_pixel];
        tmp -= A[c_pixel];
        A[c_pixel] -= (B[c_pixel] - l*C[c_pixel]);
        cuda_atomic_add(&sum[0], tmp*tmp);
    }
}
__global__ void cuda_kernel_update_momentum(XFLOAT *grads,
                                            XFLOAT *momentum,
                                            XFLOAT mu,
                                            XFLOAT l_r,
                                            int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size){
        XFLOAT tmp = momentum[pixel];
        momentum[pixel] = mu*momentum[pixel] - l_r*grads[pixel];
        grads[pixel] = tmp;
    }
}

__global__ void cuda_kernel_soft_threshold(XFLOAT *img,
                                           XFLOAT *momentum,
                                           XFLOAT *grads,
                                           int Z,
                                           int Y,
                                           int X,
                                           XFLOAT mu,
                                           XFLOAT l_r,
                                           XFLOAT alpha,
                                           XFLOAT eps,
                                           int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size){
        int kp = pixel / (Y*X);
        int ip = (pixel - kp * (Y*X))/X;
        int jp = pixel - kp*(Y*X) - ip*X;
        int hZ = Z >> 1;
        int hY = Y >> 1;
        int hX = X >> 1;
        if(kp >= hZ) kp += Z;
        if(ip >= hY) ip += Y;
        if(jp >= hX) jp += X;
        hY = Y << 1;
        hX = X << 1;
        int c_pixel = kp*hY*hX + ip*hX + jp;
        XFLOAT th = l_r*alpha/(eps+img[c_pixel]);
        XFLOAT tmp = momentum[pixel];
        XFLOAT n_momentum = mu*tmp - l_r*grads[c_pixel];
        momentum[pixel] = n_momentum;
        grads[c_pixel] = img[c_pixel];
        img[c_pixel] += n_momentum + mu*(n_momentum - tmp);
        if(img[c_pixel] < th && img[c_pixel] > -th){
            img[c_pixel] = 0.;
        } else {
            if(img[c_pixel] >= th){
                img[c_pixel] -= th;
            } else {
                img[c_pixel] += th;
            }
        }
        grads[c_pixel] -= img[c_pixel];
    }
}

__global__ void cuda_kernel_soft_threshold(XFLOAT *img,
                                           XFLOAT *grads,
                                           XFLOAT l_r,
                                           XFLOAT alpha,
                                           XFLOAT eps,
                                           int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size){
        XFLOAT th = l_r*alpha/(eps+img[pixel]);
        XFLOAT tmp = img[pixel];
        img[pixel] -=  l_r*grads[pixel];
        grads[pixel] = tmp;
        if(img[pixel] < th && img[pixel] > -th){
            img[pixel] = 0.;
        } else {
            if(img[pixel] >= th){
                img[pixel] -= th;
            } else {
                img[pixel] += th;
            }
        }
        grads[pixel] -= img[pixel];
    }
}

__global__ void cuda_kernel_soft_threshold(XFLOAT *img,
                                           XFLOAT *grads,
                                           XFLOAT l_r,
                                           XFLOAT alpha,
                                           XFLOAT eps,
                                           int X,
                                           int Y,
                                           int Z,
                                           int XX,
                                           int YY,
                                           int ZZ,
                                           int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size){
        int k = pixel/(YY*XX);
        int i = (pixel - k*YY*XX)/XX;
        int j = pixel - k*YY*XX - i*XX;
        int hZ = ZZ >> 1;
        int hY = YY >> 1;
        int hX = XX >> 1;
        int kl = k;
        int il = i;
        int jl = j;
        if(kl >= hZ){
            kl -= ZZ;
            k = kl + Z;
        }
        if(il >= hY){
            il -= YY;
            i = il + Y;
        }
        if(jl >= hX){
            jl -= XX;
            j = jl + X;
        }

        pixel = k*Y*X + i*X + j;
        XFLOAT th = l_r*alpha/(eps+img[pixel]);
        XFLOAT tmp = img[pixel];
        img[pixel] -=  l_r*grads[pixel];
        //grads[pixel] = tmp;
        if(img[pixel] < th && img[pixel] > -th){
            img[pixel] = 0.;
        } else {
            if(img[pixel] >= th){
                img[pixel] -= th;
            } else {
                img[pixel] += th;
            }
        }
        //grads[pixel] -= img[pixel];
    }
}

__global__ void cuda_kernel_soft_threshold(XFLOAT *img,
                                           XFLOAT *momentum,
                                           XFLOAT *grads,
                                           XFLOAT mu,
                                           XFLOAT l_r,
                                           XFLOAT alpha,
                                           XFLOAT eps,
                                           int X,
                                           int Y,
                                           int Z,
                                           int XX,
                                           int YY,
                                           int ZZ,
                                           int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size){
        int k = pixel/(YY*XX);
        int i = (pixel - k*YY*XX)/XX;
        int j = pixel - k*YY*XX - i*XX;
        int hZ = ZZ >> 1;
        int hY = YY >> 1;
        int hX = XX >> 1;
        int kl = k;
        int il = i;
        int jl = j;
        if(kl >= hZ){
            kl -= ZZ;
            k = kl + Z;
        }
        if(il >= hY){
            il -= YY;
            i = il + Y;
        }
        if(jl >= hX){
            jl -= XX;
            j = jl + X;
        }
        pixel = k*Y*X + i*X + j;
        XFLOAT th = l_r*alpha/(eps+img[pixel]);
        //store image first
        XFLOAT tmp = img[pixel];
        //threshold result goest to image
        momentum[pixel] -=  l_r*grads[pixel];
        //grads[pixel] = tmp;
        if(momentum[pixel] < th && momentum[pixel] > -th){
            img[pixel] = 0.;
        } else {
            if(momentum[pixel] >= th){
                img[pixel] = momentum[pixel] - th;
            } else {
                img[pixel] = momentum[pixel] + th;
            }
        }
        //mix new image with old image to get new momentum
        grads[pixel] = img[pixel] - tmp;
        momentum[pixel] = img[pixel] + mu*grads[pixel];
    }
}

__global__ void cuda_kernel_soft_threshold(XFLOAT *img,
                                           XFLOAT *momentum,
                                           XFLOAT *grads,
                                           XFLOAT *curvature,
                                           XFLOAT mu,
                                           XFLOAT l_r,
                                           XFLOAT alpha,
                                           XFLOAT eps,
                                           XFLOAT epsadam,
                                           XFLOAT mut,
                                           int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size){
        l_r /= (sqrt(curvature[pixel])+epsadam);
        XFLOAT th = l_r*alpha/(eps+img[pixel]);
        XFLOAT tmp = momentum[pixel];
        XFLOAT tmp_grad = grads[pixel];
        XFLOAT n_momentum = mu*tmp - (1. - mu)*tmp_grad;
        momentum[pixel] = n_momentum;
        grads[pixel] = img[pixel];
        img[pixel] += (mu*n_momentum + (1. - mu)*tmp_grad)*l_r/(1. - mut);
        if(img[pixel] < th && img[pixel] > -th){
            img[pixel] = 0.;
        } else {
            if(img[pixel] >= th){
                img[pixel] -= th;
            } else {
                img[pixel] += th;
            }
        }
        grads[pixel] -= img[pixel];
    }
}

__global__ void cuda_kernel_graph_grad(XFLOAT *img,
                                       XFLOAT *grads,
                                       int Y,
                                       int X,
                                       XFLOAT beta,
                                       XFLOAT eps,
                                       int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size){
        XFLOAT val = img[pixel];
        int i = pixel/X;
        int j = pixel - i*X;
        int hY = Y>>1;
        int hX = X>>1;
        XFLOAT tmp = 0.;
        int il = i;// + hY;
        int jl = j;// + hX;
        if (il >= hY) il -= Y;
        if (jl >= hX) jl -= X;
        //il -= hY;
        //jl -= hX;
        XFLOAT norm = 0.;
        XFLOAT gtmp = 0.;
        if( il < hY - 1){
            int ipp = il + 1;
            if(il < -1) ipp += Y;
            int loc = ipp*X + j;
            XFLOAT img_loc = img[loc];
            tmp += val - img_loc;
            norm += (val - img_loc)*(val - img_loc);
        }
        if( jl < hX - 1){
            int jpp = jl + 1;
            if(jl < -1) jpp += X;
            int loc = i*X + jpp;
            XFLOAT img_loc = img[loc];
            tmp += val - img_loc;
            norm += (val - img_loc)*(val - img_loc);
        }
        if(norm > eps*eps){
            tmp /= sqrt(norm);
            gtmp += tmp*beta;
        } else {
            gtmp += tmp*beta/eps;
        }
        //got the norm of il - 1, jl
        if( il > -hY ){
            int ipm = il - 1;
            if(il < 1) ipm += Y;
            val = img[ipm*X + j];
            tmp = img[pixel] - val;
            norm = tmp*tmp;
            if( jl < hX - 1){
                int jpp = jl + 1;
                if(jl < -1) jpp += X;
                int loc = ipm*X + jpp;
                XFLOAT img_loc = img[loc];
                norm += (val - img_loc)*(val - img_loc);
            }
            if(norm > eps*eps){
                tmp /= sqrt(norm);
                gtmp += tmp*beta;
            } else {
                gtmp += tmp*beta/eps;
            }
        }
        //got the norm of il, jl - 1
        //il ranges from 0, hX - 1, -hX, -1
        if( jl > -hX ){
            int jpm = jl - 1;
            if(jl < 1) jpm += X;
            val = img[i*X + jpm];
            tmp = img[pixel] - val;
            norm = tmp*tmp;
            if( il < hY - 1){
                int ipp = il + 1;
                if(il < -1) ipp += X;
                int loc = ipp*X + jpm;
                XFLOAT img_loc = img[loc];
                norm += (val - img_loc)*(val - img_loc);
            }
            if(norm > eps*eps){
                tmp /= sqrt(norm);
                gtmp += tmp*beta;
            } else {
                gtmp += tmp*beta/eps;
            }
        }
        grads[pixel] += gtmp;
        //if( ip > -hY)
        //{
        //    int ipp = ip - 1;
        //    if(ip < 1) ipp += Y;
        //    int loc = ipp*X + j;
        //    tmp += val - img[loc];
        //}
        //if( ip < hY - 1)
        //{
        //    int ipp = ip + 1;
        //    if(ip < -1) ipp += Y;
        //    int loc = ipp*X + j;
        //    tmp += val - img[loc];
        //}
        //if( jp > -hX)
        //{
        //    int jpp = jp - 1;
        //    if(jp < 1) jpp += X;
        //    int loc = i*X + jpp;
        //    tmp += val - img[loc];
        //}
        //if( jp < hX - 1)
        //{
        //    int jpp = jp + 1;
        //    if(jp < -1) jpp += X;
        //    int loc = i*X + jpp;
        //    tmp += val - img[loc];
        //}
        //grads[pixel] += tmp*beta;
    }

}

__global__ void cuda_kernel_graph_grad(XFLOAT *img,
                                       XFLOAT *grads,
                                       int Z,
                                       int Y,
                                       int X,
                                       int ZZ,
                                       int YY,
                                       int XX,
                                       XFLOAT beta,
                                       XFLOAT epslog,
                                       XFLOAT eps,
                                       int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size){
        int k = pixel/(YY*XX);
        int i = (pixel - k*YY*XX)/XX;
        int j = pixel - k*YY*XX - i*XX;
        int hZ = ZZ>>1;
        int hY = YY>>1;
        int hX = XX>>1;
        XFLOAT tmp = 0.;
        int kl = k ;//+ hZ;
        int il = i ;//+ hY;
        int jl = j ;//+ hX;
        if (kl >= hZ) {
            kl -= ZZ;
            k  += ZZ;
        }
        if (il >= hY) {
            il -= YY;
            i  += YY;
        }
        if (jl >= hX) {
            jl -= XX;
            j  += XX;
        }
        XFLOAT val = img[k*Y*X+i*X+j];
        XFLOAT norm = 0.;
        XFLOAT gtmp = 0.;
        int kpp = kl + 1;
        if(kl < -1) kpp += Z;
        int ipp = il + 1;
        if(il < -1) ipp += Y;
        int jpp = jl + 1;
        if(jl < -1) jpp += X;

        if( kl < hZ - 1){
            int loc = kpp*Y*X + i*X + j;
            XFLOAT img_loc = img[loc];
            tmp += val - img_loc;
            norm += (val - img_loc)*(val - img_loc);
        }
        if( il < hY - 1){
            int loc = k*Y*X + ipp*X + j;
            XFLOAT img_loc = img[loc];
            tmp += val - img_loc;
            norm += (val - img_loc)*(val - img_loc);
        }
        if( jl < hX - 1){
            int loc = k*Y*X + i*X + jpp;
            XFLOAT img_loc = img[loc];
            tmp += val - img_loc;
            norm += (val - img_loc)*(val - img_loc);
        }
        norm = sqrt(norm);
        if(norm > eps){
            tmp /= norm;
        } else {
            tmp /= eps;
        }
        gtmp += tmp/(norm + epslog)*beta;
        //got the norm of kl-1, il, jl
        //kl - 1 >= -hZ
        if( kl > -hZ ){
            int kpm = kl - 1;
            //kl - 1 < 0
            if(kl < 1) kpm += Z;
            XFLOAT nval = img[kpm*Y*X + i*X + j];
            tmp = val - nval;
            norm = tmp*tmp;
            //il + 1 < hY
            if( il < hY - 1){
                int loc = kpm*Y*X + ipp*X + j;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            if( jl < hX - 1){
                int loc = kpm*Y*X + i*X + jpp;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            norm = sqrt(norm);
            if(norm > eps){
                tmp /= norm;
            } else {
                tmp /= eps;
            }
            gtmp += tmp/(norm + epslog)*beta;
        }
        //got the norm of kl, il - 1, jl
        if( il > -hY ){
            norm = 0.;
            int ipm = il - 1;
            if(il < 1) ipm += Y;
            XFLOAT nval = img[k*Y*X + ipm*X + j];
            tmp = val - nval;
            norm = tmp*tmp;
            if( kl < hZ - 1){
                int loc = kpp*Y*X + ipm*X + j;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            if( jl < hX - 1){
                int loc = k*Y*X + ipm*X + jpp;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            norm = sqrt(norm);
            if(norm > eps){
                tmp /= norm;
            } else {
                tmp /= eps;
            }
            gtmp += tmp*beta/(norm + epslog);
        }
        //got the norm of kl, il, jl - 1
        if( jl > -hX ){
            int jpm = jl - 1;
            if(jl < 1) jpm += X;
            XFLOAT nval = img[k*Y*X + i*X + jpm];
            tmp = val - nval;
            norm = tmp*tmp;
            if( kl < hZ - 1){
                int loc = kpp*Y*X + i*X + jpm;
                norm += (nval - img[loc])*(nval - img[loc]);
            }
            if( il < hY - 1){
                int loc = k*Y*X + ipp*X + jpm;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            norm = sqrt(norm);
            if(norm > eps){
                tmp /= norm;
            } else {
                tmp /= eps;
            }
            gtmp += tmp/(norm + epslog)*beta;
        }
        //grads[pixel] += gtmp;
        grads[k*Y*X+i*X+j] += gtmp;
    }
}

/*__global__ void cuda_kernel_graph_grad_shared(XFLOAT *img,
                                       XFLOAT *grads,
                                       int Z,
                                       int Y,
                                       int X,
                                       XFLOAT beta,
                                       XFLOAT epslog,
                                       XFLOAT eps,
                                       int image_size)
{
    __shared__ XFLOAT img_shared[TILE_SIZE][TILE_SIZE][PENCIL_SIZE];
    int iy = threadIdx.y + blockDim.y*blockIdx.y;
    int iz = threadIdx.z + blockDim.z*blockIdx.z;
    if(iy < Y && iz < Z){
        for(int i = 0; i < ceilf(XFLOAT(X)/blockDim.x); i++) {
            int ix = threadIdx.x + i*blockDim.x + blockDim.x*blockIdx.x;
            int pixel = ix + iy*X + iz*Y*X;
            img_shared[threadIdx.z][threadIdx.y][threadIdx.x] = img[pixel];
            if(threadIdx.x == PENCIL_SIZE - 2){
                img_shared[threadIdx.z][threadIdx.y][threadIdx.x+1] = img[pixel + 1];
            }
            if(threadIdx.y == TILE_SIZE - 2) {
                pixel = ix + (iy + 1)*X + iz*Y*X;
                img_shared[threadIdx.z][threadIdx.y+1][threadIdx.x] = img[pixel];
            }
            if(threadIdx.z == TILE_SIZE - 2) {
                pixel = ix + iy*X + (iz + 1)*Y*X;
                img_shared[threadIdx.z+1][threadIdx.y][threadIdx.x] = img[pixel];
            }
            if(threadIdx.x == X - 1){
                pixel = iy*X + iz*Y*X;
                img_shared[threadIdx.z][threadIdx.y][threadIdx.x+1] = img[pixel];
            }
            if(threadIdx.y == Y - 1){
                pixel = ix + iz*Y*X;
                img_shared[threadIdx.z][threadIdx.y+1][threadIdx.x] = img[pixel];
            }
            if(threadIdx.z == Z - 1){
                pixel = ix + iy*X;
                img_shared[threadIdx.z+1][threadIdx.y+1][threadIdx.x] = img[pixel];
            }
            __syncthreads();
            XFLOAT norm = 0.;
            XFLOAT gtmp = 0.;
            XFLOAT diffx = img_shared[threadIdx.z][threadIdx.y][threadIdx.x] - img_shared[threadIdx.z][threadIdx.y][threadIdx.x+1];
            XFLOAT diffy = img_shared[threadIdx.z][threadIdx.y][threadIdx.x] - img_shared[threadIdx.z][threadIdx.y+1][threadIdx.x];
            XFLOAT diffz = img_shared[threadIdx.z][threadIdx.y][threadIdx.x] - img_shared[threadIdx.z+1][threadIdx.y][threadIdx.x];
            norm = diffx*diffx + diffy*diffy + diffz*diffz;
            norm = sqrt(norm);
            gtmp = diffx + diffy + diffz;
            XFLOAT snorm = norm;
            if(norm > eps){
                snorm = eps;
            }
            norm += epslog;
            gtmp = gtmp/snorm/norm*beta;
            pixel = ix + iy*X + iz*Y*X;
            atomicAdd(grad + pixel, gtmp);
            pixel = (ix + 1)%X + iy*X + iz*Y*X;
            gtmp = -diffx/snorm/norm*beta;
            atomicAdd(grad + pixel, gtmp);
            gtmp = -diffy/snorm/norm*beta;
            pixel = ix + ((iy+1) % Y)*X + iz*Y*X;
            atomicAdd(grad + pixel, gtmp);
            gtmp = -diffz/snorm/norm*beta;
            pixel = ix + iy*X + ((iz+1) % Z)*Y*X;
            atomicAdd(grad + pixel, gtmp);
        }

    }

}*/

__global__ void cuda_kernel_graph_grad(XFLOAT *img,
                                       XFLOAT *grads,
                                       int Z,
                                       int Y,
                                       int X,
                                       XFLOAT beta,
                                       XFLOAT epslog,
                                       XFLOAT eps,
                                       int image_size)
{
    int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    if(pixel < image_size){
        XFLOAT val = img[pixel];
        int k = pixel/(Y*X);
        int i = (pixel - k*Y*X)/X;
        int j = pixel - k*Y*X - i*X;
        int hZ = Z>>1;
        int hY = Y>>1;
        int hX = X>>1;
        XFLOAT tmp = 0.;
        int kl = k ;//+ hZ;
        int il = i ;//+ hY;
        int jl = j ;//+ hX;
        if (kl >= hZ) kl -= Z;
        if (il >= hY) il -= Y;
        if (jl >= hX) jl -= X;
        //kl -= hZ;
        //il -= hY;
        //jl -= hX;
        XFLOAT norm = 0.;
        XFLOAT gtmp = 0.;
        int kpp = kl + 1;
        if(kl < -1) kpp += Z;
        int ipp = il + 1;
        if(il < -1) ipp += Y;
        int jpp = jl + 1;
        if(jl < -1) jpp += X;

        if( kl < hZ - 1){
            int loc = kpp*Y*X + i*X + j;
            XFLOAT img_loc = img[loc];
            tmp += val - img_loc;
            norm += (val - img_loc)*(val - img_loc);
        }
        if( il < hY - 1){
            int loc = k*Y*X + ipp*X + j;
            XFLOAT img_loc = img[loc];
            tmp += val - img_loc;
            norm += (val - img_loc)*(val - img_loc);
        }
        if( jl < hX - 1){
            int loc = k*Y*X + i*X + jpp;
            XFLOAT img_loc = img[loc];
            tmp += val - img_loc;
            norm += (val - img_loc)*(val - img_loc);
        }
        norm = sqrt(norm);
        if(norm > eps){
            tmp /= norm;
        } else {
            tmp /= eps;
        }
        gtmp += tmp/(norm + epslog)*beta;
        //got the norm of kl-1, il, jl
        if( kl > -hZ ){
            int kpm = kl - 1;
            if(kl < 1) kpm += Z;
            XFLOAT nval = img[kpm*Y*X + i*X + j];
            tmp = val - nval;
            norm = tmp*tmp;
            if( il < hY - 1){
                int loc = kpm*Y*X + ipp*X + j;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            if( jl < hX - 1){
                int loc = kpm*Y*X + i*X + jpp;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            norm = sqrt(norm);
            if(norm > eps){
                tmp /= norm;
            } else {
                tmp /= eps;
            }
            gtmp += tmp/(norm + epslog)*beta;
        }
        //got the norm of kl, il - 1, jl
        if( il > -hY ){
            norm = 0.;
            int ipm = il - 1;
            if(il < 1) ipm += Y;
            XFLOAT nval = img[k*Y*X + ipm*X + j];
            tmp = val - nval;
            norm = tmp*tmp;
            if( kl < hZ - 1){
                int loc = kpp*Y*X + ipm*X + j;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            if( jl < hX - 1){
                int loc = k*Y*X + ipm*X + jpp;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            norm = sqrt(norm);
            if(norm > eps){
                tmp /= norm;
            } else {
                tmp /= eps;
            }
            gtmp += tmp*beta/(norm + epslog);
        }
        //got the norm of kl, il, jl - 1
        if( jl > -hX ){
            int jpm = jl - 1;
            if(jl < 1) jpm += X;
            XFLOAT nval = img[k*Y*X + i*X + jpm];
            tmp = val - nval;
            norm = tmp*tmp;
            if( kl < hZ - 1){
                int loc = kpp*Y*X + i*X + jpm;
                norm += (nval - img[loc])*(nval - img[loc]);
            }
            if( il < hY - 1){
                int loc = k*Y*X + ipp*X + jpm;
                XFLOAT img_loc = img[loc];
                norm += (nval - img_loc)*(nval - img_loc);
            }
            norm = sqrt(norm);
            if(norm > eps){
                tmp /= norm;
            } else {
                tmp /= eps;
            }
            gtmp += tmp/(norm + epslog)*beta;
        }
        grads[pixel] += gtmp;
        //if( kp > -hZ)
        //{
        //    int kpp = kp - 1;
        //    if(kp < 1) kpp += Z;
        //    int loc = kpp*Y*X + i*X + j;
        //    tmp += val - img[loc];
        //}
        //if( kp < hZ - 1)
        //{
        //    int kpp = kp + 1;
        //    if(kp < -1) kpp += Z;
        //    int loc = kpp*Y*X + i*X + j;
        //    tmp += val - img[loc];
        //}
        //if( ip > -hY)
        //{
        //    int ipp = ip - 1;
        //    if(ip < 1) ipp += Y;
        //    int loc = k*Y*X + ipp*X + j;
        //    tmp += val - img[loc];
        //}
        //if( ip < hY - 1)
        //{
        //    int ipp = ip + 1;
        //    if(ip < -1) ipp += Y;
        //    int loc = k*Y*X + ipp*X + j;
        //    tmp += val - img[loc];
        //}
        //if( jp > -hX)
        //{
        //    int jpp = jp - 1;
        //    if(jp < 1) jpp += X;
        //    int loc = k*Y*X + i*X + jpp;
        //    tmp += val - img[loc];
        //}
        //if( jp < hX - 1)
        //{
        //    int jpp = jp + 1;
        //    if(jp < -1) jpp += X;
        //    int loc = k*Y*X + i*X + jpp;
        //    tmp += val - img[loc];
        //}
        //grads[pixel] += tmp*beta;
    }

}

__global__ void cuda_kernel_finalizeMstddev( XFLOAT *Mstddev,
											 XFLOAT *aux,
											 XFLOAT S,
											 int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
	{
		XFLOAT temp = Mstddev[pixel] + S * aux[pixel];
		if(temp > 0)
			Mstddev[pixel] = sqrt(temp);
		else
			Mstddev[pixel] = 0;
	}
}

__global__ void cuda_kernel_square(
		XFLOAT *A,
		int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
		A[pixel] = A[pixel]*A[pixel];
}

__global__ void cuda_kernel_square(
		XFLOAT *A,
        XFLOAT *B,
        XFLOAT beta,
		int image_size)
{
	int pixel = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	if(pixel<image_size)
		B[pixel] = (1. - beta)*A[pixel]*A[pixel] + beta*B[pixel];
}
